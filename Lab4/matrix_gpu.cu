#include "hip/hip_runtime.h"
// Simple CUDA example by Ingemar Ragnemalm 2009. Simplest possible?
// Assigns every element in an array with its index.

// nvcc simple.cu -L /usr/local/cuda/lib -lcudart -o simple

#include <stdio.h>
#include <math.h>

const int GRID_SIZE = 1;
const int BLOCK_SIZE = 2048;
const int N = GRID_SIZE*BLOCK_SIZE;

__global__
void simple(float* in1, float* in2, float* out)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;
    int id = idx * N + idy;
    out[id] = in1[id] + in2[id];
}

int main()
{

    float* a = new float[N*N];
    float* b = new float[N*N];
    float* c = new float[N*N];

    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            a[i+j*N] = 10 + i;
            b[i+j*N] = (float)j / N;
        }
    }

    float* a_cuda;
    float* b_cuda;
    float* c_cuda;
    
    hipEvent_t event_start;
    hipEvent_t event_end;
    hipEventCreate(&event_start);
    hipEventCreate(&event_end);

    const int size = N*N*sizeof(float);

    hipMalloc( (void**)&a_cuda, size );
    hipMalloc( (void**)&b_cuda, size );
    hipMalloc( (void**)&c_cuda, size );

    hipEventRecord(event_start, 0);
    hipMemcpy(a_cuda, a, size, hipMemcpyHostToDevice);
    hipMemcpy(b_cuda, b, size, hipMemcpyHostToDevice);
    hipMemcpy(c_cuda, c, size, hipMemcpyHostToDevice);

    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid(GRID_SIZE, GRID_SIZE);
    simple<<<dimGrid, dimBlock>>>(a_cuda, b_cuda, c_cuda);

    hipDeviceSynchronize();
    hipEventRecord(event_end, 0);
    hipEventSynchronize(event_end);

    hipMemcpy(c, c_cuda, size, hipMemcpyDeviceToHost);

    float myVerySpecial_Time_not_t;
    hipEventElapsedTime(&myVerySpecial_Time_not_t, event_start, event_end);

    printf("Time elapsed: %f\n", myVerySpecial_Time_not_t);

    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
        {
            printf("%0.2f ", c[i+j*N]);
        }
        printf("\n");
    }

    hipFree(a_cuda);
    hipFree(b_cuda);
    hipFree(c_cuda);

    printf("done\n");
    return EXIT_SUCCESS;
}
