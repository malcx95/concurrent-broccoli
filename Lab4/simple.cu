#include "hip/hip_runtime.h"
// Simple CUDA example by Ingemar Ragnemalm 2009. Simplest possible?
// Assigns every element in an array with its index.

// nvcc simple.cu -L /usr/local/cuda/lib -lcudart -o simple

#include <stdio.h>
#include <math.h>


const int N = 16; 
const int blocksize = 16; 

__global__
void simple(float *c) 
{
    c[threadIdx.x] = sqrt(c[threadIdx.x]);
}

int main()
{
    float *c = new float[N];
    float *cd;
    const int size = N*sizeof(float);

    printf("ON CPU:\n");
    for (int i = 0; i < N; ++i) {
        c[i] = i;
        printf("%f ", sqrt(c[i]));
    }

    hipMalloc( (void**)&cd, size );
    hipMemcpy(cd, c, size, hipMemcpyHostToDevice);
    dim3 dimBlock( blocksize, 1 );
    dim3 dimGrid( 1, 1 );
    simple<<<dimGrid, dimBlock>>>(cd);
    hipDeviceSynchronize();
    hipMemcpy( c, cd, size, hipMemcpyDeviceToHost ); 
    hipFree( cd );

    printf("ON gPU:\n");
    for (int i = 0; i < N; i++)
        printf("%f ", c[i]);
    printf("\n");
    delete[] c;
    printf("done\n");
    return EXIT_SUCCESS;
}
